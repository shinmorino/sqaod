#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <>
#include "DeviceBipartiteGraphBatchSearch.h"
#include "cub_iterator.cuh"

using namespace sqaod_cuda;

template<class real>
DeviceBipartiteGraphBatchSearch<real>::DeviceBipartiteGraphBatchSearch() {
    N0_ = N1_ = -1;
}


template<class real>
void DeviceBipartiteGraphBatchSearch<real>::assignDevice(Device &device, DeviceStream *devStream) {
    devStream_ = devStream;
    devFormulas_.assignDevice(device, devStream_);
    devCopy_.assignDevice(device, devStream_);
    devAlloc_ = device.objectAllocator();
}

template<class real>
void DeviceBipartiteGraphBatchSearch<real>::deallocate() {
    devAlloc_->deallocate(d_b0_);
    devAlloc_->deallocate(d_b1_);
    devAlloc_->deallocate(d_W_);
    devAlloc_->deallocate(d_bitsMat0_);
    devAlloc_->deallocate(d_bitsMat1_);
    devAlloc_->deallocate(d_Ebatch_);
    devAlloc_->deallocate(d_minXPairs_);

    HostObjectAllocator halloc;
    halloc.deallocate(h_nMinXPairs_);
    halloc.deallocate(h_Emin_);
}


template<class real>
void DeviceBipartiteGraphBatchSearch<real>::
setQUBO(const HostVector &b0, const HostVector &b1,
        const HostMatrix &W,
        sq::SizeType tileSize0, sq::SizeType tileSize1) {
    if (N0_ != -1)
        deallocate();

    N0_ = b0.size;
    N1_ = b1.size;
    devCopy_(&d_b0_, b0);
    devCopy_(&d_b1_, b1);
    devCopy_(&d_W_, W);
    tileSize0_ = tileSize0;
    tileSize1_ = tileSize1;
    minXPairsSize_ = tileSize0 * tileSize1;
    devAlloc_->allocate(&d_bitsMat0_, tileSize0, W.cols);
    devAlloc_->allocate(&d_bitsMat1_, tileSize1, W.rows);
    devAlloc_->allocate(&d_Ebatch_, tileSize1, tileSize0);
    devAlloc_->allocate(&d_minXPairs_, tileSize1 * tileSize0);

    HostObjectAllocator halloc;
    halloc.allocate(&h_nMinXPairs_);
    halloc.allocate(&h_Emin_);
}


template<class real>
void DeviceBipartiteGraphBatchSearch<real>::
calculate_E(sq::PackedBitSet xBegin0, sq::PackedBitSet xEnd0,
            sq::PackedBitSet xBegin1, sq::PackedBitSet xEnd1) {
    xBegin0_ = xBegin0;
    xBegin1_ = xBegin1;
    sq::SizeType nBatch0 = sq::SizeType(xEnd0 - xBegin0);
    sq::SizeType nBatch1 = sq::SizeType(xEnd1 - xBegin1);
    abortIf(tileSize0_ < nBatch0,
            "nBatch0 is too large, tileSize0=%d, nBatch0=%d", int(tileSize0_), int(nBatch0));
    abortIf(tileSize0_ < nBatch0,
            "nBatch1 is too large, tileSize1=%d, nBatch1=%d", int(tileSize1_), int(nBatch1));
    /* FIXME: use stream if effective */
    generateBitsSequence(&d_bitsMat0_, xBegin0, xEnd0);
    generateBitsSequence(&d_bitsMat1_, xBegin1, xEnd1);
    devFormulas_.calculate_E_2d(&d_Ebatch_, d_b0_, d_b1_, d_W_, d_bitsMat0_, d_bitsMat1_);
    devFormulas_.devMath.min(&h_Emin_, d_Ebatch_);
}


template<class real>
void DeviceBipartiteGraphBatchSearch<real>::partition_minXPairs(bool append) {
    assert(d_Ebatch_.dim() == sq::Dim(tileSize1_, tileSize0_));
    if (!append) {
        d_minXPairs_.size = 0;
        select(d_minXPairs_.d_data, h_nMinXPairs_.d_data,
               xBegin0_, xBegin1_, *h_Emin_.d_data,
               d_Ebatch_.d_data, d_Ebatch_.stride, tileSize0_, tileSize1_);
        synchronize();
        d_minXPairs_.size = *h_nMinXPairs_.d_data; /* sync field */
    }
    else if (d_minXPairs_.size < minXPairsSize_) {
        /* append */
        select(&d_minXPairs_.d_data[d_minXPairs_.size], h_nMinXPairs_.d_data,
               xBegin0_, xBegin1_, *h_Emin_.d_data,
               d_Ebatch_.d_data, d_Ebatch_.stride, tileSize0_, tileSize1_);
        synchronize();
        d_minXPairs_.size += *h_nMinXPairs_.d_data; /* sync field */
    }
}

template<class real>
void DeviceBipartiteGraphBatchSearch<real>::synchronize() {
    devStream_->synchronize();
}


template<class real>
__global__ static
void generateBitsSequenceKernel(real *d_data, int stride, int N,
                                sq::SizeType nSeqs, sq::PackedBitSet xBegin) {
    sq::IdxType seqIdx = blockDim.y * blockIdx.x + threadIdx.y;
    if ((seqIdx < nSeqs) && (threadIdx.x < N)) {
        sq::PackedBitSet bits = xBegin + seqIdx;
        bool bitSet = bits & (1ull << (N - 1 - threadIdx.x));
        d_data[seqIdx * stride + threadIdx.x] = bitSet ? real(1) : real(0);
    }
}


template<class real> void DeviceBipartiteGraphBatchSearch<real>::
generateBitsSequence(DeviceMatrix *bitsSequences,
                     sq::PackedBitSet xBegin, sq::PackedBitSet xEnd) {
    sq::SizeType N = bitsSequences->cols;
    sq::SizeType stride = bitsSequences->stride;
    dim3 blockDim, gridDim;
    blockDim.x = roundUp(N, 32); /* Packed bits <= 63 bits. */
    blockDim.y = 128 / blockDim.x; /* 2 or 4, sequences per block. */
    sq::SizeType nSeqs = sq::SizeType(xEnd - xBegin);
    gridDim.x = divru((unsigned int)(xEnd - xBegin), blockDim.y);
    generateBitsSequenceKernel<<<gridDim, blockDim, 0, devStream_->getCudaStream()>>>
            (bitsSequences->d_data, stride, N, nSeqs, xBegin);
    DEBUG_SYNC;
}


namespace {

struct SelectInputIterator {
    __device__ __forceinline__
    SelectInputIterator(sq::PackedBitSetPair _xPairOffset, int _tileSize0)
            : xPairOffset(_xPairOffset), tileSize0(_tileSize0) { }
    __host__
    SelectInputIterator(int _tileSize0) { 
        xPairOffset.bits0 = 0;
        xPairOffset.bits1 = 0;
        tileSize0 = _tileSize0;
    }

    __device__ __forceinline__
    sq::PackedBitSetPair operator[](int idx) const {
        sq::PackedBitSetPair pair;
        pair.bits0 = xPairOffset.bits0 + (idx % tileSize0);
        pair.bits1 = xPairOffset.bits1 + (idx / tileSize0);
        return pair;
    }
    __device__ __forceinline__
    SelectInputIterator operator+(int idx) {
        sq::PackedBitSetPair pair;
        pair.bits0 = xPairOffset.bits0 + (idx % tileSize0);
        pair.bits1 = xPairOffset.bits1 + (idx / tileSize0);
        return SelectInputIterator(pair, tileSize0);
    }
    sq::PackedBitSetPair xPairOffset;
    int tileSize0;
};


struct SelectOutput {
    __device__ __forceinline__
    SelectOutput(sq::PackedBitSet _xBegin0, sq::PackedBitSet _xBegin1, sq::PackedBitSetPair &_d_out)
            : xBegin0(_xBegin0), xBegin1(_xBegin1), d_out(_d_out) { }
    __device__ __forceinline__
    void operator=(sq::PackedBitSetPair &v) const {
        sq::PackedBitSetPair pair;
        pair.bits0 = v.bits0 + xBegin0;
        pair.bits1 = v.bits1 + xBegin1;
        d_out = pair;
    }
    sq::PackedBitSet xBegin0;
    sq::PackedBitSet xBegin1;
    sq::PackedBitSetPair &d_out;
};

struct SelectOutputIterator {
    SelectOutputIterator(sq::PackedBitSet _xBegin0, sq::PackedBitSet _xBegin1,
                         sq::PackedBitSetPair *_d_out) : xBegin0(_xBegin0), xBegin1(_xBegin1),
                                                         d_out(_d_out) { }
    __device__ __forceinline__
    SelectOutput operator[](unsigned int idx) const {
        return SelectOutput(xBegin0, xBegin1, d_out[idx]);
    }
    sq::PackedBitSet xBegin0;
    sq::PackedBitSet xBegin1;
    sq::PackedBitSetPair *d_out;
};


template<class real> struct SelectOp {
    SelectOp(real _val, const real *_d_vals, int _stride)
            : val(_val), d_vals(_d_vals), stride(_stride) { }
    __device__ __forceinline__
    bool operator()(const sq::PackedBitSetPair &idx) const {
        return val == d_vals[idx.bits1 * stride + idx.bits0];
    }
    real val;
    const real *d_vals;
    int stride;
};

}

namespace std {
template<>
struct iterator_traits<SelectInputIterator> : sqaod_cuda::base_iterator_traits<sq::PackedBitSetPair> { };
template<>
struct iterator_traits<SelectOutputIterator> : sqaod_cuda::base_iterator_traits<sq::PackedBitSetPair> { };
template<class real>
struct iterator_traits<SelectOp<real> > : sqaod_cuda::base_iterator_traits<real> { };

}


template<class real> void DeviceBipartiteGraphBatchSearch<real>::
select(sq::PackedBitSetPair *d_out, sq::SizeType *d_nOut,
       sq::PackedBitSet xBegin0, sq::PackedBitSet xBegin1, 
       real val, const real *d_vals, sq::SizeType valsStride, sq::SizeType nIn0, sq::SizeType nIn1) {
    SelectInputIterator in(tileSize0_);

    SelectOutputIterator out(xBegin0, xBegin1, d_out);
    SelectOp<real> selectOp(val, d_vals, valsStride);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    sq::SizeType nIn = nIn0 * nIn1;
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes,
                          in, out, d_nOut, nIn, selectOp, devStream_->getCudaStream(), CUB_DEBUG);
    // Allocate temporary storage
    d_temp_storage = devStream_->allocate(temp_storage_bytes);
    // Run selection
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes,
                          in, out, d_nOut, nIn, selectOp, devStream_->getCudaStream(), CUB_DEBUG);
}



template class sqaod_cuda::DeviceBipartiteGraphBatchSearch<double>;
template class sqaod_cuda::DeviceBipartiteGraphBatchSearch<float>;




// template<class real>
// void BGFuncs<real>::batchSearch(real *E, PackedBitSetPairArray *xPairs,
//                                 const EigenDeviceMatrix &b0, const EigenDeviceMatrix &b1, const EigenDeviceMatrix &W,
//                                 PackedBitSet xBegin0, PackedBitSet xEnd0,
//                                 PackedBitSet xBegin1, PackedBitSet xEnd1) {
//     int nBatch0 = int(xEnd0 - xBegin0);
//     int nBatch1 = int(xEnd1 - xBegin1);

//     real Emin = *E;
//     int N0 = W.cols();
//     int N1 = W.rows();
//     EigenDeviceMatrix eBitsSeq0(nBatch0, N0);
//     EigenDeviceMatrix eBitsSeq1(nBatch1, N1);

//     createBitsSequence(eBitsSeq0.data(), N0, xBegin0, xEnd0);
//     createBitsSequence(eBitsSeq1.data(), N1, xBegin1, xEnd1);
    
//     EigenDeviceMatrix eEBatch = eBitsSeq1 * (W * eBitsSeq0.transpose());
//     eEBatch.rowwise() += (b0 * eBitsSeq0.transpose()).row(0);
//     eEBatch.colwise() += (b1 * eBitsSeq1.transpose()).transpose().col(0);
    
//     /* FIXME: Parallelize */
//     for (int idx1 = 0; idx1 < nBatch1; ++idx1) {
//         for (int idx0 = 0; idx0 < nBatch0; ++idx0) {
//             real Etmp = eEBatch(idx1, idx0);
//             if (Etmp > Emin) {
//                 continue;
//             }
//             else if (Etmp == Emin) {
//                 xPairs->push_back(PackedBitSetPairArray::value_type(xBegin0 + idx0, xBegin1 + idx1));
//             }
//             else {
//                 Emin = Etmp;
//                 xPairs->clear();
//                 xPairs->push_back(PackedBitSetPairArray::value_type(xBegin0 + idx0, xBegin1 + idx1));
//             }
//         }
//     }
//     *E = Emin;
// }
    

